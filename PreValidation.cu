﻿#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_cuda()
{
	printf("Hello cuda: %d\n", threadIdx.x);
}


void start_hello()
{
	hello_cuda<<<1, 4>>>();
}
